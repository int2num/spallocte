#include "hip/hip_runtime.h"

/**
 * Copyright 1993-2012 NVIDIA Corporation.  All rights reserved.
 *
 * Please refer to the NVIDIA end user license agreement (EULA) associated
 * with this source code for terms and conditions that govern your use of
 * this software. Any use, reproduction, disclosure, or distribution of
 * this software and related documentation outside the terms of the EULA
 * is strictly prohibited.
 */
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include"pathalg.h"
static const int WORK_SIZE = 128;
__global__ void bellmanHigh(edge *edges,int*d,int*m,int edgesize)
{
	int i = threadIdx.x + blockIdx.x*blockDim.x;
	if (i >=edgesize)return;
	int head = edges[i].s;
	int tail = edges[i].t;
	int w=edges[i].w;
	int val = d[head]+w;
	if (d[tail] >val){
		*m = 1;
		d[tail] = val;
	}
};
__global__ void color(edge *edges,int* d,int*pre,int edgesize)
{
	int i = threadIdx.x + blockIdx.x*blockDim.x;
	if (i >=edgesize)return;
	int head = edges[i].s;
	int tail = edges[i].t;
	int w=edges[i].w;
	int val=d[head]+w;
	if (d[tail]==val&&d[tail]<INT_MAX/3)
		pre[tail]=i;
};
void parallelor::allocate(int s,vector<edge>&edges,int nodenum){
	hipMalloc(&dev_edges, sizeof(edge)*edges.size());
	hipMalloc((void**)&dev_d,nodenum*sizeof(int));
	hipMalloc((void**)&dev_pre,nodenum*sizeof(int));
	hipMalloc((void**)&dev_m,sizeof(int));
	m=new int;
	*m=0;
	d=new int[nodenum],pre=new int[nodenum];
	memset(pre,-1,sizeof(int)*nodenum);
	for(int i=0;i<nodenum;i++)
		d[i]=INT_MAX/2;
	d[s]=0;
	aedges=new edge[edges.size()];
	for(int i=0;i<edges.size();i++)
		aedges[i]=edges[i];
	hipMemcpy(dev_edges,aedges,edges.size()* sizeof(edge),hipMemcpyHostToDevice);
	hipMemcpy(dev_m,m,sizeof(int),hipMemcpyHostToDevice);
	hipMemcpy(dev_d,d,sizeof(int)*nodenum,hipMemcpyHostToDevice);
	hipMemcpy(dev_pre,pre,sizeof(int)*nodenum,hipMemcpyHostToDevice);
};
void parallelor::dellocate(){
	delete[]d;
	delete[]pre;
	delete[]aedges;
	delete m;
	hipFree(dev_edges);
	hipFree(dev_m);
	hipFree(dev_d);
	hipFree(dev_pre);
};
parallelor::parallelor(){};
vector<int> parallelor:: routalg(int s,int t,vector<edge>&edges,int nodenum)
{
	dijkstor plo=dijkstor();
	allocate(s,edges,nodenum);
	int zero=0;
	int ck=0;
	do{
		if(ck%5==0)
			hipMemcpy(dev_m, &zero, sizeof(int), hipMemcpyHostToDevice);
		bellmanHigh << <(edges.size()/WORK_SIZE)+1, WORK_SIZE >> >(dev_edges,dev_d,dev_m,edges.size());
		if(ck%5==0)
			hipMemcpy(m, dev_m, sizeof(int), hipMemcpyDeviceToHost);
		ck++;
	}
	while(*m>0);
	color << <(edges.size()/WORK_SIZE)+1, WORK_SIZE >> >(dev_edges,dev_d,dev_pre,edges.size());
	hipMemcpy(pre,dev_pre,sizeof(int)*nodenum,hipMemcpyDeviceToHost);
	hipMemcpy(d,dev_d,sizeof(int)*nodenum,hipMemcpyDeviceToHost);
	vector<int>vpree;
	for(int i=0;i<nodenum;i++)
		vpree.push_back(pre[i]);
	dellocate();
	return getrout(s,t,edges,vpree);

};
